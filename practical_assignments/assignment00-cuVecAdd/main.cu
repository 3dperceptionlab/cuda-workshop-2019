#include "hip/hip_runtime.h"
/// ---------------------------------------------------------------------------
/// CUDA Workshop 2019
/// Universidad de Alicante
/// Pr�ctica 0 - Suma de Vectores
/// C�digo preparado por: Albert Garc�a <agarcia@dtic.ua.es>
///                       Sergio Orts <sorts@ua.es>
/// ---------------------------------------------------------------------------

#include <iostream>
// Cabecera necesaria para las rutinas del runtime de CUDA (hipFree, hipMalloc...)
#include <hip/hip_runtime.h>
// Cabecera necesaria para variables y tipos de CUDA...
#include <>

/// Kernel para suma de vectores.
/// Este kernel computar� la suma de dos vectores de forma que cada
/// hilo ser� responsable de sumar un elemento de dichos vectores.
__global__ 
void suma_vectores(
	const float *cpA, 
	const float *cpB, 
	float *pC, 
	const int cNumElements)
{
	/// === PASO 2 ============================================================
	/// Define los �ndices del elemento a ser sumado por cada hilo empleando las
	/// variables de CUDA: threadIdx, blockIdx y blockDim.
	/// TODO:
	int idx_ = ???;

	/// Suma las dos posiciones en el vector de salida, cada hilo debe computar
	/// el c�lculo de un elemento.
	/// TODO:

	/// === FIN PASO 2 ========================================================
}

/// Kernel para suma de vectores con stride.
/// Este kernel computar� la suma de dos vectores de forma que cada
/// hilo ser� responsable de sumar varios elementos de dichos vectores.
__global__
void suma_vectores_strided(
	const float *cpA,
	const float *cpB,
	float *pC,
	const int cNumElements)
{
	/// === PASO 4 ============================================================
	/// Modifica el kernel anterior para que se puedan sumar vectores de un
	/// tama�o muy grande. Recuerda cambiar los par�metros de invocaci�n y
	/// llamar a este kernel en lugar de al anterior.
	int idx_ = ???;

	/// Sumar las posiciones adecuadas en el vector de salida, cada hilo debe
	/// computar m�s de un elemento.
	/// TODO:

	/// === FINAL PASO 4 ======================================================
}

int main(void)
{
	// Elegimos la GPU a utilizar, en este caso la 0
	hipSetDevice(0);

	// Calculamos el tama�o en bytes del vector
	/// === PASO 3 ============================================================
	/// Modifica el n�mero de elementos a sumar.
  const int kNumElements = 25600;
	/// === FIN PASO 3 ========================================================
  size_t vector_size_bytes_ = kNumElements * sizeof(float);
	std::cout << "[Vector addition of " << kNumElements << " elements]\n";

  // Reservamos memoria para los vectores en el HOST
  float *h_A_ = (float *)malloc(vector_size_bytes_);
  float *h_B_ = (float *)malloc(vector_size_bytes_);
  float *h_C_ = (float *)malloc(vector_size_bytes_);

  // Comprobamos que las reservas se han efectuado correctamente
  if (h_A_ == NULL || h_B_ == NULL || h_C_ == NULL)
  {
		std::cerr << "Failed to allocate host vectors!\n";
		getchar();
    exit(-1);
  }

  // Inicializamos los vectores en el HOST con valores arbitrarios
  for (int i = 0; i < kNumElements; ++i)
  {
		h_A_[i] = rand()/(float)RAND_MAX;
		h_B_[i] = rand()/(float)RAND_MAX;
  }

	// Reservamos memoria para los vectores en el DEVICE
  float *d_A_ = NULL;
	float *d_B_ = NULL;
	float *d_C_ = NULL;

  hipMalloc((void **)&d_A_, vector_size_bytes_);
	hipMalloc((void **)&d_B_, vector_size_bytes_);
	hipMalloc((void **)&d_C_, vector_size_bytes_);

  // Copiamos los vectores A y B de HOST a DEVICE
	std::cout << "Copy input data from the host memory to the CUDA device\n";

  hipMemcpy(d_A_, h_A_, vector_size_bytes_, hipMemcpyHostToDevice);
  hipMemcpy(d_B_, h_B_, vector_size_bytes_, hipMemcpyHostToDevice);

  // Lanzamos el kernel de suma de vectores y comprobamos errores
	/// === PASO 1 ============================================================
	/// Establece los par�metros de invocaci�n del kernel e inv�calo.
  int threads_per_block_ = 256;
	int blocks_per_grid_ = ???;
	/// === PASO 3 ============================================================
	/// Modifica el c�lculo del tama�o del grid para que se puedan sumar
	/// vectores de un tama�o arbitrario.
	/// blocks_per_grid_ = ???;
	/// === FIN PASO 3 ========================================================
	/// === PASO 4 ============================================================
	/// Establece los par�metros de invocaci�n del kernel modificado.
	/// blocks_per_grid_ = ???;
	/// === FIN PASO 4 ========================================================

	dim3 block(threads_per_block_, 1, 1);
	dim3 grid(blocks_per_grid_, 1, 1);

	std::cout << "CUDA kernel launch with " << blocks_per_grid_ << " blocks of " << threads_per_block_ << " threads\n";
  suma_vectores<<<???, ???>>>(???);
	hipError_t err_ = hipGetLastError();

  if (err_ != hipSuccess)
  {
		std::cerr << "Failed to launch sumaVectores kernel (error code " << hipGetErrorString(err_) << ")!\n";
		getchar();
    exit(-1);
  }
	/// === FIN PASO 1 ========================================================

  // Copiamos el vector resultante del DEVICE al HOST
	std::cout << "Copy output data from the CUDA device to the host memory\n";

  hipMemcpy(h_C_, d_C_, vector_size_bytes_, hipMemcpyDeviceToHost);

  // Verificamos el resultado
  for (int i = 0; i < kNumElements; ++i)
  {
		// Dado que utilizamos floats las comparaciones de igualdad fallar�an
		// por el orden de las operaciones por lo que utilizamos una comparaci�n
		// con un umbral 1e-5
    if (fabs(h_A_[i] + h_B_[i] - h_C_[i]) > 1e-5)
    {
			std::cerr << "Result verification failed at element " << i << "!\n";
			getchar();
      exit(-1);
    }
  }

	std::cout << "Test PASSED\n";

  // Liberamos la memoria en el DEVICE
  hipFree(d_A_);
  hipFree(d_B_);
	hipFree(d_C_);

  // Liberamos la memoria en el HOST
  free(h_A_);
  free(h_B_);
  free(h_C_);

	// Reiniciamos el dispositivo
	// hipDeviceReset hace que el driver limpie todo estado actual. Aunque no es
	// una operaci�n obligatoria, es una buena pr�ctica. Adem�s, es necesaria si
	// estamos realiando profiling de la aplicaci�n.
  hipDeviceReset();

	// Finalizamos el programa
	std::cout << "Done\n";
	getchar();
  return 0;
}

