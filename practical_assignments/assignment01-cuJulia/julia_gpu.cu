#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#include "./common/book.h"
#include "./common/cpu_bitmap.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <>

#define DIM 1000

/* retorna "a - b" en segundos - RUTINA TIEMPOS WINDOWS */
double performancecounter_diff(LARGE_INTEGER *a, LARGE_INTEGER *b)
{
  LARGE_INTEGER freq;
  QueryPerformanceFrequency(&freq);
  return (double)(a->QuadPart - b->QuadPart) / (double)freq.QuadPart;
}

struct hipComplex {
    float   r;
    float   i;
    __host__ __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __host__ __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __host__ __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __host__ __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

__host__ __device__ int julia( int x, int y ) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernel( unsigned char *ptr ) {	
	
	// COMPLETAR!
	//
	int x = //TODO
	int y = //TODO
	int offset = y * (blockDim.x*gridDim.x) + x;

    // now calculate the value at that position
    int juliaValue = julia( x, y );
    ptr[offset*4 + 0] = 255 * juliaValue;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

void kernel_CPU( unsigned char *ptr ){
    for (int y=0; y<DIM; y++) {
        for (int x=0; x<DIM; x++) {
            int offset = x + y * DIM;

            int juliaValue = julia( x, y );
            ptr[offset*4 + 0] = 255 * juliaValue;
            ptr[offset*4 + 1] = 0;
            ptr[offset*4 + 2] = 0;
            ptr[offset*4 + 3] = 255;
        }
    }
 }

// globals needed by the update routine
struct DataBlock {
    unsigned char   *dev_bitmap;
};

int main( void ) {
    DataBlock   data;
    CPUBitmap bitmap( DIM, DIM, &data );
    unsigned char    *dev_bitmap;

	// Declaramos los eventos y los inicializamos
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );

    HANDLE_ERROR( hipMalloc( (void**)&dev_bitmap, bitmap.image_size() ) );
    data.dev_bitmap = dev_bitmap;
    
	// COMPLETAR
	// TODO
	dim3    grid();
	dim3	block();
	kernel<<< >>>( dev_bitmap );
    
	HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), dev_bitmap,
                              bitmap.image_size(),
                              hipMemcpyDeviceToHost ) );
    
	// Detenemos los eventos y mostramos los tiempos
  HANDLE_ERROR( hipEventRecord( stop, 0 ) );
  HANDLE_ERROR( hipEventSynchronize( stop ) );
  float   elapsedTime;
  HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
  printf( "Time consumido generacion fractal GPU:  %3.1f ms\n", elapsedTime );
	HANDLE_ERROR( hipEventDestroy( start ) );
  HANDLE_ERROR( hipEventDestroy( stop ) );
	HANDLE_ERROR( hipFree( dev_bitmap ) );
      
	// Calculo CPU
	LARGE_INTEGER t_ini, t_fin;
	CPUBitmap bitmap2( DIM, DIM );
	unsigned char *ptr = bitmap2.get_ptr();

	QueryPerformanceCounter(&t_ini);	
  kernel_CPU( ptr );
  QueryPerformanceCounter(&t_fin);

  double cpu_runtime = performancecounter_diff(&t_fin, &t_ini);
  printf( "Time consumido generacion fractal CPU:  %3.1f ms\n", cpu_runtime*1000);

  bitmap.display_and_exit();
}

